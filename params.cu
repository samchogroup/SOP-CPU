#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstring>
#include <cstdlib>
#include <cstdio>
#include <math.h>
#include <iostream>
#include <fstream>
#include "params.h"
#include "io.h"
#include "global.h"

void set_params(int icmd)
{

  using namespace std;
  char oline[1024];
  int iopt;

  if( !strcmp(opt[opt_ptr[icmd]],"dynamics") ) { // set the type of simulation
    if( !strcmp(opt[opt_ptr[icmd]+1],"underdamped") ) {
      sim_type = 1; // low-friction limit for sampling
      h = 2.5e-3;
      halfh = h/2.0;
      a1 = h*(1.0-zeta*halfh);
      a2 = h*halfh;
      a3 = (1.0-h*zeta/2.0+(h*zeta)*(h*zeta)/4.0)/h;
      a4 = halfh*(1.0-h*zeta/2.0);
    } else if( !strcmp(opt[opt_ptr[icmd]+1],"overdamped") ) {
      sim_type = 2; // hi-friction limit for kinetics
      h = 0.02;
      a5 = h/zeta;
    }
  } else if( !strcmp(opt[opt_ptr[icmd]],"temp") ) { // set the temperature
    set_temp(atof(opt[opt_ptr[icmd]+1]));

  } else if( !strcmp(opt[opt_ptr[icmd]],"nstep") ) { // # of steps
    nstep = atof(opt[opt_ptr[icmd]+1]);
  } else if( !strcmp(opt[opt_ptr[icmd]],"istep_restart") ) { // where to restart from
    istep_restart = atof(opt[opt_ptr[icmd]+1]);
  } else if( !strcmp(opt[opt_ptr[icmd]],"nup") ) { // # of steps before an update
    nup = atoi(opt[opt_ptr[icmd]+1]);

  } else if( !strcmp(opt[opt_ptr[icmd]],"run") ) { // set current run
    run = atoi((opt[opt_ptr[icmd]+1]));
    generator.set_seed(-100-run);

  } else if( !strcmp(opt[opt_ptr[icmd]],"ufname") ) { // set update file name
    strcpy(ufname,opt[opt_ptr[icmd]+1]);
  } else if( !strcmp(opt[opt_ptr[icmd]],"rcfname") ) { // set restart coordinate file name
    strcpy(rcfname,opt[opt_ptr[icmd]+1]);
  } else if( !strcmp(opt[opt_ptr[icmd]],"cfname") ) { // set save coordinate file name
    strcpy(cfname,opt[opt_ptr[icmd]+1]);
  } else if( !strcmp(opt[opt_ptr[icmd]],"rgenfname") ) { // set random generator file name
    generator.set_fname(opt[opt_ptr[icmd]+1]);
  } else if( !strcmp(opt[opt_ptr[icmd]],"unccfname") ) { // set save coordinate file name
    strcpy(unccfname,opt[opt_ptr[icmd]+1]);
  } else if( !strcmp(opt[opt_ptr[icmd]],"vfname") ) { // set save velocity file name
    strcpy(vfname,opt[opt_ptr[icmd]+1]);
  } else if( !strcmp(opt[opt_ptr[icmd]],"binfname") ) { // set save trajectory file name
    strcpy(binfname,opt[opt_ptr[icmd]+1]);
  } else if( !strcmp(opt[opt_ptr[icmd]],"uncbinfname") ) { // set save trajectory file name
    strcpy(uncbinfname,opt[opt_ptr[icmd]+1]);

  } else if( !strcmp(opt[opt_ptr[icmd]],"cutofftype") ) { // neighbor list on or off?
    if( !strcmp(opt[opt_ptr[icmd]+1],"neighborlist" ) ) { neighborlist = 1; }
    else if( !strcmp(opt[opt_ptr[icmd]+1],"celllist" ) ) { celllist = 1; }
    else { }

  } else if( !strcmp(opt[opt_ptr[icmd]],"nnlup") ) { // neighbor / cell list update frequency
    nnlup = atoi(opt[opt_ptr[icmd]+1]);

  } else if( !strcmp(opt[opt_ptr[icmd]],"boxl") ) { // box length for pbc
    boxl = atof(opt[opt_ptr[icmd]+1]);
  } else if( !strcmp(opt[opt_ptr[icmd]],"ncell") ) { // number of cells along box length
    ncell = atof(opt[opt_ptr[icmd]+1]);
    lcell = boxl / ncell;

  } else if( !strcmp(opt[opt_ptr[icmd]],"restart") ) { // restart on or off?
    if( !strcmp(opt[opt_ptr[icmd]+1],"on" ) ) { restart = 1; }
    else { restart = 0; }
  } else if( !strcmp(opt[opt_ptr[icmd]],"rgen_restart") ) { // restart the generator?
    if( !strcmp(opt[opt_ptr[icmd]+1],"on" ) ) { rgen_restart = 1; }
    else { rgen_restart = 0; }
  } else if( !strcmp(opt[opt_ptr[icmd]],"t_step") ) {
    h = atof((opt[opt_ptr[icmd]+1]));
    halfh = h/2.0;
    a1 = h*(1.0-zeta*halfh);
    a2 = h*halfh;
    a3 = (1.0-h*zeta/2.0+(h*zeta)*(h*zeta)/4.0)/h;
    a4 = halfh*(1.0-h*zeta/2.0);
    a5 = h/zeta;
  } else if( !strcmp(opt[opt_ptr[icmd]],"zeta") ) { // friction coefficient
    if (sim_type == 1) h = 2.5e-3; else if (sim_type == 2) h = 0.02;
    zeta = atof((opt[opt_ptr[icmd]+1]));
    a1 = h*(1.0-zeta*halfh);
    a3 = (1.0-h*zeta/2.0+(h*zeta)*(h*zeta)/4.0)/h;
    a4 = halfh*(1.0-h*zeta/2.0);
    a5 = h/zeta;
  } else {};

}

void set_temp(double temp)
{
  using namespace std;

  T = temp;
}

void load(int icmd)
{
  using namespace std;

  ifstream in;
  char line[2048];
  char* tokPtr;
  int test;
  int test1,test2;
  int ncon_tot;
  int icon_att, icon_rep;
  int i,j,k,l;
  int ibead,jbead;
  int itype,jtype;
  double real_phi, ideal_phi;
  double r_ij;
  int istate;

  if( !strcmp(opt[opt_ptr[icmd]],"bonds") ) { // load bonds
    cout << "[Reading in bonds...]" << endl;
    in.clear();
    in.open(opt[opt_ptr[icmd]+1],ios::in); // open file
    in.getline(line,2048);
    tokPtr = strtok(line," ");
    tokPtr = strtok(NULL," ");
    nbnd = atoi(tokPtr); // read in number of bonds
    init_bonds(nbnd);
    for( int i=0; i<nbnd; i++ ) {
      in.getline(line,2048);
      tokPtr = strtok(line," ");
      ibead_bnd[i] = atoi(tokPtr); // first bead index
      tokPtr = strtok(NULL," ");
      jbead_bnd[i] = atoi(tokPtr); // second bead index
      tokPtr = strtok(NULL," ");
      pdb_dist[i] = atof(tokPtr); // equilibrium distance (angstrom)
    }
    in.close(); // close file
    cout << "[Finished reading bonds (" << nbnd <<")]" << endl;
  } else if(!strcmp(opt[opt_ptr[icmd]],"angles")) { // load angles
    cout << "[Reading in angles...]" << endl;
    in.clear();
    in.open(opt[opt_ptr[icmd]+1],ios::in);
    in.getline(line,2048);
    tokPtr = strtok(line," ");
    tokPtr = strtok(NULL," ");
    nang = atoi(tokPtr); // read in number of angles
    init_angles(nang);
    for( int i=0; i<nang; i++ ) {
      in.getline(line,2048);
      tokPtr = strtok(line," ");
      ibead_ang[i] = atoi(tokPtr); // first bead index
      tokPtr = strtok(NULL," ");
      jbead_ang[i] = atoi(tokPtr); // second bead index
      tokPtr = strtok(NULL," ");
      kbead_ang[i] = atoi(tokPtr); // third bead index
      tokPtr = strtok(NULL," ");
      pdb_ang[i] = atof(tokPtr); // equilibrium angle (radians) ; SOP -> dist between bead i,i+2
    }
    in.close();
    cout << "[Finished reading angles (" << nang <<")]" << endl;
  } else if(!strcmp(opt[opt_ptr[icmd]],"vdw")) { // load rna-rna vdw
    cout << "[Reading in VDW interactions...]" << endl;
    in.clear();
    in.open(opt[opt_ptr[icmd]+1],ios::in);
    in.getline(line,2048);
    tokPtr = strtok(line," ");
    tokPtr = strtok(NULL," ");
    ncon_att = atoi(tokPtr);
    tokPtr = strtok(NULL," ");
    tokPtr = strtok(NULL," ");
    ncon_rep = atoi(tokPtr);
    init_lj(ncon_att,ncon_rep);
    ncon_tot = ncon_att + ncon_rep;
    icon_att = 0;
    icon_rep = 0;
    for( int i=0; i<ncon_tot; i++ ) {
      in.getline(line,2048);
      tokPtr = strtok(line," ");
      ibead = atoi(tokPtr);
      tokPtr = strtok(NULL," ");
      jbead = atoi(tokPtr);
      tokPtr = strtok(NULL," ");
      r_ij = atof(tokPtr);
      tokPtr = strtok(NULL," ");
      itype = atoi(tokPtr);
      tokPtr = strtok(NULL," ");
      jtype = atoi(tokPtr);
      if (r_ij < rcut_nat[itype][jtype]) {
	ibead_lj_nat[icon_att] = ibead;
	jbead_lj_nat[icon_att] = jbead;
	itype_lj_nat[icon_att] = itype;
	jtype_lj_nat[icon_att] = jtype;
	lj_nat_pdb_dist[icon_att] = r_ij;
	lj_nat_pdb_dist2[icon_att] = r_ij*r_ij;
	lj_nat_pdb_dist6[icon_att] = lj_nat_pdb_dist2[icon_att]*
	  lj_nat_pdb_dist2[icon_att]*lj_nat_pdb_dist2[icon_att];
	lj_nat_pdb_dist12[icon_att] = lj_nat_pdb_dist6[icon_att]*
	  lj_nat_pdb_dist6[icon_att];
	icon_att++;

	ibead_pair_list_att[nil_att] = ibead;
	jbead_pair_list_att[nil_att] = jbead;
	itype_pair_list_att[nil_att] = itype;
	jtype_pair_list_att[nil_att] = jtype;
	pl_lj_nat_pdb_dist[nil_att] = r_ij;
	pl_lj_nat_pdb_dist2[nil_att] = lj_nat_pdb_dist2[icon_att];
	pl_lj_nat_pdb_dist6[nil_att] = lj_nat_pdb_dist6[icon_att];
	pl_lj_nat_pdb_dist12[nil_att] = lj_nat_pdb_dist12[icon_att];
	nil_att++;
      } else {
	ibead_lj_non_nat[icon_rep] = ibead;
	jbead_lj_non_nat[icon_rep] = jbead;
	itype_lj_non_nat[icon_rep] = itype;
	jtype_lj_non_nat[icon_rep] = jtype;
	icon_rep++;

	ibead_pair_list_rep[nil_rep] = ibead;
	jbead_pair_list_rep[nil_rep] = jbead;
	itype_pair_list_rep[nil_rep] = itype;
	jtype_pair_list_rep[nil_rep] = jtype;
	nil_rep++;
      }
    }
    in.close();
    cout << "[Finished reading VDW interactions (" << icon_att << "/" << icon_rep <<")]" << endl;
  } else if(!strcmp(opt[opt_ptr[icmd]],"init")) { // load init coordinates
    cout << "[Reading in initial coordinates...]" << endl;
    in.clear();
    in.open(opt[opt_ptr[icmd]+1],ios::in);
    in.getline(line,2048);
    tokPtr = strtok(line," ");
    tokPtr = strtok(NULL," ");
    nbead = atoi(tokPtr); // read in number of beads
    init_pos(nbead);
    for( int i=0; i<nbead; i++ ) {
      in.getline(line,2048);
      tokPtr = strtok(line," ");
      tokPtr = strtok(NULL," ");
      pos[i].x = atof(tokPtr);
      unc_pos[i].x = pos[i].x;
      tokPtr = strtok(NULL," ");
      pos[i].y = atof(tokPtr);
      unc_pos[i].y = pos[i].y;
      tokPtr = strtok(NULL," ");
      pos[i].z = atof(tokPtr);
      unc_pos[i].z = pos[i].z;
    }
    in.close();
    cout << "[Finished reading initial coordinates (" << nbead << ")]" << endl;
  }
}

void alloc_arrays()
{
  using namespace std;

  // bonds

  k_bnd = 20.0;
  R0 = 2.0; // = 0.4*a
  R0sq = R0*R0;
  e_bnd_coeff = k_bnd*R0sq/2.0; // SOP model
  nbnd = 1529;
  ibead_bnd = new int[nbnd];
  jbead_bnd = new int[nbnd];
  pdb_dist = new double[nbnd];
  bnds_allocated = 1;

  // angles

  k_ang = 20.0;
  e_ang_coeff = k_ang/2.0;
  nang = 1528;
  ibead_ang = new int[nang];
  jbead_ang = new int[nang];
  kbead_ang = new int[nang];
  pdb_ang = new double[nang];
  angs_allocated = 1;

  sigma_ss = 3.5; // = 0.76*a
  sigma_ss6 = pow(sigma_ss,6.0);
  epsilon_ss = 1.0;
  e_ang_ss_coeff = epsilon_ss*sigma_ss6;
  f_ang_ss_coeff = 6.0*e_ang_ss_coeff;

  // rna-rna vdw

  ncon_att = 8996;
  ncon_rep = 1157632;
  // neighbor list
  nnl_att = 0;
  nnl_rep = 0;
  // pair list
  nil_att = 0;
  nil_rep = 0;

  for (int i = 0; i < 3; i++) {
    for (int j = 0; j < 3; j++) {
      sigma_rep2[i][j] = sigma_rep[i][j] * sigma_rep[i][j];
      sigma_rep6[i][j] = sigma_rep2[i][j] * sigma_rep2[i][j] * sigma_rep2[i][j];
      sigma_rep12[i][j] = sigma_rep6[i][j] * sigma_rep6[i][j];
    }
  }

  ibead_lj_nat = new int[ncon_att];
  jbead_lj_nat = new int[ncon_att];
  itype_lj_nat = new int[ncon_att];
  jtype_lj_nat = new int[ncon_att];
  lj_nat_pdb_dist = new double[ncon_att];
  lj_nat_pdb_dist2 = new double[ncon_att];
  lj_nat_pdb_dist6 = new double[ncon_att];
  lj_nat_pdb_dist12 = new double[ncon_att];
  ibead_lj_non_nat = new int[ncon_rep];
  jbead_lj_non_nat = new int[ncon_rep];
  itype_lj_non_nat = new int[ncon_rep];
  jtype_lj_non_nat = new int[ncon_rep];

  ibead_neighbor_list_att = new int[ncon_att];
  jbead_neighbor_list_att = new int[ncon_att];
  itype_neighbor_list_att = new int[ncon_att];
  jtype_neighbor_list_att = new int[ncon_att];
  nl_lj_nat_pdb_dist = new double[ncon_att];
  nl_lj_nat_pdb_dist2 = new double[ncon_att];
  nl_lj_nat_pdb_dist6 = new double[ncon_att];
  nl_lj_nat_pdb_dist12 = new double[ncon_att];
  ibead_neighbor_list_rep = new int[ncon_rep];
  jbead_neighbor_list_rep = new int[ncon_rep];
  itype_neighbor_list_rep = new int[ncon_rep];
  jtype_neighbor_list_rep = new int[ncon_rep];

  ibead_pair_list_att = new int[ncon_att];
  jbead_pair_list_att = new int[ncon_att];
  itype_pair_list_att = new int[ncon_att];
  jtype_pair_list_att = new int[ncon_att];
  pl_lj_nat_pdb_dist = new double[ncon_att];
  pl_lj_nat_pdb_dist2 = new double[ncon_att];
  pl_lj_nat_pdb_dist6 = new double[ncon_att];
  pl_lj_nat_pdb_dist12 = new double[ncon_att];
  ibead_pair_list_rep = new int[ncon_rep];
  jbead_pair_list_rep = new int[ncon_rep];
  itype_pair_list_rep = new int[ncon_rep];
  jtype_pair_list_rep = new int[ncon_rep];

  lj_rna_rna_allocated = 1;

  // coordinates

  nbead = 1530;
  pos = new float3[nbead];
  unc_pos = new float3[nbead];
  vel = new float3[nbead];
  force = new float3[nbead];
  rna_base = new int [nbead];
  rna_phosphate = new int [nbead];
  pos_allocated = 1;
  unc_pos_allocated = 1;
  vel_allocated = 1;
  force_allocated = 1;
  rna_base_allocated = 1;
  rna_phosphate_allocated = 1;

  // miscellaneous run parameters

  run = 1;
  generator.set_seed(-100-run);
  T = 0.6; // kcal/mol

  neighborlist = 0; // neighbor list cutoff method?
  celllist = 0; // cell list cutoff method?
  boxl = 500.0;
  ncell = 55.0;
  lcell = boxl / ncell;
  zeta = 5.0e-2; // 0.05*tau^{-1} = friction coeff
  nstep = 5e7;
  nup = 1000;
  nnlup = 50; // neighbor list update frequency
  h = 2.5e-3;
  halfh = h/2.0;
  a1 = h*(1.0-zeta*halfh);
  a2 = h*halfh;
  a3 = (1.0-h*zeta/2.0+(h*zeta)*(h*zeta)/4.0)/h;
  a4 = halfh*(1.0-h*zeta/2.0);
  a5 = h/zeta;
  strcpy(ufname,"update.out");
  strcpy(rcfname,"restart_c.dat");
  strcpy(cfname,"coord.out");
  strcpy(unccfname,"unccoord.out");
  strcpy(vfname,"veloc.out");
  strcpy(binfname,"traj.bin");
  strcpy(uncbinfname,"traj_uncorrected.bin");
}

void release_bonds()
{
  using namespace std;

  delete [] ibead_bnd;
  delete [] jbead_bnd;
  delete [] pdb_dist;
  bnds_allocated = 0;
}

void init_bonds(int numbonds)
{
  using namespace std;

  nbnd = numbonds;
  ibead_bnd = new int[numbonds];
  jbead_bnd = new int[numbonds];
  pdb_dist = new double[numbonds];
  bnds_allocated = 1;
}

void release_angles()
{
  using namespace std;

  delete [] ibead_ang;
  delete [] jbead_ang;
  delete [] kbead_ang;
  delete [] pdb_ang;
  angs_allocated = 0;

}

void init_angles(int numangs)
{
  using namespace std;

  nang = numangs;
  ibead_ang = new int[numangs];
  jbead_ang = new int[numangs];
  kbead_ang = new int[numangs];
  pdb_ang = new double[numangs];
  angs_allocated = 1;

}

void release_lj()
{
  using namespace std;

  delete [] ibead_lj_nat;
  delete [] jbead_lj_nat;
  delete [] itype_lj_nat;
  delete [] jtype_lj_nat;
  delete [] lj_nat_pdb_dist;
  delete [] lj_nat_pdb_dist2;
  delete [] lj_nat_pdb_dist6;
  delete [] lj_nat_pdb_dist12;
  delete [] ibead_lj_non_nat;
  delete [] jbead_lj_non_nat;
  delete [] itype_lj_non_nat;
  delete [] jtype_lj_non_nat;

  delete [] ibead_neighbor_list_att;
  delete [] jbead_neighbor_list_att;
  delete [] itype_neighbor_list_att;
  delete [] jtype_neighbor_list_att;
  delete [] nl_lj_nat_pdb_dist;
  delete [] nl_lj_nat_pdb_dist2;
  delete [] nl_lj_nat_pdb_dist6;
  delete [] nl_lj_nat_pdb_dist12;
  delete [] ibead_neighbor_list_rep;
  delete [] jbead_neighbor_list_rep;
  delete [] itype_neighbor_list_rep;
  delete [] jtype_neighbor_list_rep;

  // pair list
  delete [] ibead_pair_list_att;
  delete [] jbead_pair_list_att;
  delete [] itype_pair_list_att;
  delete [] jtype_pair_list_att;
  delete [] pl_lj_nat_pdb_dist;
  delete [] pl_lj_nat_pdb_dist2;
  delete [] pl_lj_nat_pdb_dist6;
  delete [] pl_lj_nat_pdb_dist12;
  delete [] ibead_pair_list_rep;
  delete [] jbead_pair_list_rep;
  delete [] itype_pair_list_rep;
  delete [] jtype_pair_list_rep;

  lj_rna_rna_allocated = 0;

}

void init_lj(int numatt, int numrep )
{
  using namespace std;

  ncon_att = numatt;
  ncon_rep = numrep;
  ibead_lj_nat = new int[numatt];
  jbead_lj_nat = new int[numatt];
  itype_lj_nat = new int[numatt];
  jtype_lj_nat = new int[numatt];
  lj_nat_pdb_dist = new double[numatt];
  lj_nat_pdb_dist2 = new double[numatt];
  lj_nat_pdb_dist6 = new double[numatt];
  lj_nat_pdb_dist12 = new double[numatt];
  ibead_lj_non_nat = new int[numrep];
  jbead_lj_non_nat = new int[numrep];
  itype_lj_non_nat = new int[numrep];
  jtype_lj_non_nat = new int[numrep];

  ibead_neighbor_list_att = new int[numatt];
  jbead_neighbor_list_att = new int[numatt];
  itype_neighbor_list_att = new int[numatt];
  jtype_neighbor_list_att = new int[numatt];
  nl_lj_nat_pdb_dist = new double[numatt];
  nl_lj_nat_pdb_dist2 = new double[numatt];
  nl_lj_nat_pdb_dist6 = new double[numatt];
  nl_lj_nat_pdb_dist12 = new double[numatt];
  ibead_neighbor_list_rep = new int[numrep];
  jbead_neighbor_list_rep = new int[numrep];
  itype_neighbor_list_rep = new int[numrep];
  jtype_neighbor_list_rep = new int[numrep];

  ibead_pair_list_att = new int[numatt];
  jbead_pair_list_att = new int[numatt];
  itype_pair_list_att = new int[numatt];
  jtype_pair_list_att = new int[numatt];
  pl_lj_nat_pdb_dist = new double[numatt];
  pl_lj_nat_pdb_dist2 = new double[numatt];
  pl_lj_nat_pdb_dist6 = new double[numatt];
  pl_lj_nat_pdb_dist12 = new double[numatt];
  ibead_pair_list_rep = new int[numrep];
  jbead_pair_list_rep = new int[numrep];
  itype_pair_list_rep = new int[numrep];
  jtype_pair_list_rep = new int[numrep];

  lj_rna_rna_allocated = 1;

}

void init_pos(int nbead)
{
  using namespace std;

  unc_pos = new float3[nbead];
  pos = new float3[nbead];

  vel = new float3[nbead];
  force = new float3[nbead];

  pos_allocated = 1;
  unc_pos_allocated = 1;
  vel_allocated = 1;
  force_allocated = 1;
}

void release_pos()
{
  using namespace std;

  delete [] unc_pos;
  delete [] pos;

  delete [] vel;
  delete [] force;

  pos_allocated = 0;
  unc_pos_allocated = 0;
  vel_allocated = 0;
  force_allocated = 0;
}
