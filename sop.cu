#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cstring>
#include <math.h>
#include <cstdlib>
#include <ctime>
#include <cstdio>
#include <unistd.h>
#include "sop.h"
#include "random_generator.h"
#include "energy.h"
#include "io.h"
#include "params.h"
#include "neighbor_list.h"
#include "cell_list.h"

int main(int argc,char* argv[])
{

  using namespace std;

  if( argc<2 ) {
    cerr << "Usage: " << argv[0] <<  " < input_file >" << endl;
    exit(-1);
  }
  time_t tm0 = time(0); // wall time at this point
  cout << "CURRENT TIME IS: " << ctime(&tm0);
  if( getcwd(pathname,MAXPATHLEN)==NULL ) {
    cerr << "PROBLEM GETTING PATH" << endl;
  } else {
    cout << "CURRENT WORKING DIRECTORY: " << pathname << endl;
  }

  alloc_arrays(); // allocates certain arrays and initializes some variables
  read_input(argv[1]); // read input file

  clock_t ck0 = clock(); // clock ticks at this point
  ex_cmds(); // perform commands (simulation)

  // time stats
  time_t tm1 = time(0);
  clock_t ck1 = clock();
  cout << "+-------------------+" << endl;
  cout << "| Simulation Stats: |" << endl;
  cout << "+-------------------+" << endl;
  cout << "Wall Time              : " << difftime(tm1,tm0) << " sec" << endl;
  cout << "Total Computation Time : " << float(ck1-ck0)/CLOCKS_PER_SEC << " sec" << endl;
  cout << "Computation Rate       : " << float(ck1-ck0)/CLOCKS_PER_SEC/nstep << " sec / timestep" << endl;
  cout << "CURRENT TIME IS        : " << ctime(&tm1);

  return 0;

}

void ex_cmds()
{

  using namespace std;

  char oline[1024];
  int iopt;

  for( int i=1; i<=ncmd; i++ ) {
     // read data
     if( !strcmp(cmd[i],"load") ) { load(i); }
     // set parameters
     else if( !strcmp(cmd[i],"set") ) { set_params(i); }
     // run simulation
     else if( !strcmp(cmd[i],"run") ) { simulation_ctrl(); }
     // ???
     else {};
  }

}

void simulation_ctrl()
{
  using namespace std;

  switch( sim_type ) {
  case 1:
    underdamped_ctrl();
    break;
  case 2:
    overdamped_ctrl();
    break;
  default:
    cerr << "UNRECOGNIZED SIM_TYPE!" << endl;
    exit(-1);
  }
}

void underdamped_ctrl()
{
  using namespace std;

  char oline[2048];
  double istep = 1.0;
  int iup = 1;
  int inlup = 1;
  ofstream out(ufname,ios::out|ios::app);
  static int first_time = 1;

  coord* incr = new coord[nbead+1];

  if( (!restart)&&first_time ) { // zero out the velocities and forces
    for( int i=0; i<nbead; i++ ) {
      vel[i].x = 0.0;
      vel[i].y = 0.0;
      vel[i].z = 0.0;
      force[i].x = 0.0;
      force[i].y = 0.0;
      force[i].z = 0.0;
    }
  }

  print_sim_params();

  if (neighborlist == 1) {
    update_neighbor_list();
    update_pair_list();
  } else if (celllist == 1) {
    update_cell_list();
    update_pair_list();
  }

  set_potential();
  set_forces();

  char line[2048];

  if( restart ) {
    load_coords(cfname,unccfname);
    load_vels(vfname);
    istep = istep_restart + 1.0;
  }

  if( rgen_restart ) {
    generator.restart();
  }

  if( first_time ) {

    energy_eval();
    force_eval();

  }

  if( binsave ) {
    if( (first_time)&&(!rgen_restart) ) {
      record_traj(binfname,uncbinfname);
    }
    while( istep <= nstep ) {

      // compute pair separation list
      if ((inlup % nnlup) == 0) {
        if (neighborlist == 1) {
          update_neighbor_list();
        } else if (celllist == 1) {
          update_cell_list();
        }
	//	fprintf(stderr, "(%.0lf) neighbor list: (%d/%d)\n", istep, nnl_att, nnl_rep);
        inlup = 0;
      }
      inlup++;

      if (neighborlist == 1 || celllist == 1) {
        update_pair_list();
//	fprintf(stderr, "(%.0lf) pair list: (%d/%d)\n", istep, nil_att, nil_rep);
      }

      underdamped_iteration(incr);
      if( !(iup%nup) ) { // updates
	energy_eval();
	calculate_observables(incr);
        sprintf(oline,"%.0lf %f %f %f %f %f %f %f %d %f",
                istep,T,kinT,e_bnd,e_ang_ss,e_vdw_rr,rna_etot,
                Q,contct_nat,rgsq);
	out << oline << endl;
	iup = 0;
	record_traj(binfname,uncbinfname);
	save_coords(cfname,unccfname);
	save_vels(vfname);
	generator.save_state();
      }
      istep += 1.0;
      iup++;

    }
    out.close();
  }

  if( first_time ) first_time = 0;

  delete [] incr;

  return;
}

void calculate_observables(coord* increment)
{

  using namespace std;

  char oline[1024];
  double dx,dy,dz,d;
  static const double tol = 1.0; // tolerance for chi distances
  static const double chinorm = (double(nbead*nbead)-5.0*double(nbead)+6.0)/2.0;
  double sumvsq;
  int nchi;
  int ibead, jbead;
  int itype, jtype;
  float r_ij;
  char line[2048];

  // chi, contct_nat, contct_tot, Q

  contct_nat = 0;
  for( int i=0; i<ncon_att; i++ ) {

    ibead = ibead_lj_nat[i];
    jbead = jbead_lj_nat[i];
    r_ij = lj_nat_pdb_dist[i];
    itype = itype_lj_nat[i];
    jtype = jtype_lj_nat[i];

    dx = unc_pos[ibead].x-unc_pos[jbead].x;
    dy = unc_pos[ibead].y-unc_pos[jbead].y;
    dz = unc_pos[ibead].z-unc_pos[jbead].z;

    dx -= boxl*rnd(dx/boxl);
    dy -= boxl*rnd(dy/boxl);
    dz -= boxl*rnd(dz/boxl);

    d = sqrt( dx*dx+dy*dy+dz*dz );
    if(d/r_ij < 1.25) {
      contct_nat++;
    }
  }
  Q = double(contct_nat)/ncon_att;


  // rgsq

  rgsq = 0.0;
  for( int i=0; i<nbead-1; i++ ) {
    for( int j=i+1; j<=nbead; j++ ) {
      dx = unc_pos[i].x-unc_pos[j].x;
      dy = unc_pos[i].y-unc_pos[j].y;
      dz = unc_pos[i].z-unc_pos[j].z;
      dx -= boxl*rnd(dx/boxl);
      dy -= boxl*rnd(dy/boxl);
      dz -= boxl*rnd(dz/boxl);

      rgsq += (dx*dx+dy*dy+dz*dz);
    }
  }
  rgsq /= double(nbead*nbead);

  // kinT

  if( sim_type == 1 ) {
    sumvsq = 0.0;
    for( int i=0; i<nbead; i++ ) {
      sumvsq += vel[i].x*vel[i].x
	+ vel[i].y*vel[i].y
	+ vel[i].z*vel[i].z;
    }
    kinT = sumvsq/(3.0*double(nbead));
  } else if( sim_type == 2 ) {
    sumvsq = 0.0;
    for( int i=0; i<nbead; i++ ) {
      sumvsq += increment[i].x*increment[i].x +
	increment[i].y*increment[i].y +
	increment[i].z*increment[i].z;
    }
    sumvsq *= zeta/(2.0*h);
    kinT = sumvsq/(3.0*double(nbead));
  } else {}
}

void underdamped_iteration(coord* incr)
{
  using namespace std;

  static const double eps = 1.0e-5;

  for( int i=0; i<nbead; i++ ) {

    // compute position increments

    incr[i].x = a1*vel[i].x + a2*force[i].x;
    incr[i].y = a1*vel[i].y + a2*force[i].y;
    incr[i].z = a1*vel[i].z + a2*force[i].z;

    // update bead positions

    pos[i].x += incr[i].x;
    pos[i].y += incr[i].y;
    pos[i].z += incr[i].z;

    pos[i].x -= boxl*rnd(pos[i].x/boxl);
    pos[i].y -= boxl*rnd(pos[i].y/boxl);
    pos[i].z -= boxl*rnd(pos[i].z/boxl);

    unc_pos[i].x += incr[i].x;
    unc_pos[i].y += incr[i].y;
    unc_pos[i].z += incr[i].z;

  }

  // force_update

  force_eval();

  if( T < eps ) return; // don't update velocities for steepest descent

  // update_velocities

  for( int i=0; i<nbead; i++ ) {

    // compute velocity increments

    vel[i].x = a3*incr[i].x + a4*force[i].x;
    vel[i].y = a3*incr[i].y + a4*force[i].y;
    vel[i].z = a3*incr[i].z + a4*force[i].z;

  }
}

void overdamped_iteration(coord* incr)
{
   using namespace std;

   for( int i=0; i<nbead; i++ ) {

      // compute position increments

      incr[i].x = a5*force[i].x;
      incr[i].y = a5*force[i].y;
      incr[i].z = a5*force[i].z;

      // update bead positions

      unc_pos[i].x += incr[i].x;
      unc_pos[i].y += incr[i].y;
      unc_pos[i].z += incr[i].z;

      pos[i].x += incr[i].x;
      pos[i].y += incr[i].y;
      pos[i].z += incr[i].z;

      pos[i].x -= boxl*rnd(pos[i].x/boxl);
      pos[i].y -= boxl*rnd(pos[i].y/boxl);
      pos[i].z -= boxl*rnd(pos[i].z/boxl);

   }

   // force_update

   force_eval();

}

void overdamped_ctrl()
{

  using namespace std;

  char oline[2048];
  double istep = 1.0;
  int iup = 1;
  ofstream out(ufname,ios::out|ios::app);
  static int first_time = 1;

  coord* incr = new coord[nbead+1];

  if( (!restart)&&first_time ) { // zero out the velocities and forces
    for( int i=0; i<nbead; i++ ) {
      vel[i].x = 0.0;
      vel[i].y = 0.0;
      vel[i].z = 0.0;
      force[i].x = 0.0;
      force[i].y = 0.0;
      force[i].z = 0.0;
    }
  }

  print_sim_params();

  if (neighborlist == 1) {
    update_neighbor_list();
    update_pair_list();
  } else if (celllist == 1) {
    update_cell_list();
    update_pair_list();
  }

  set_potential();
  set_forces();

  char line[2048];

  if( restart ) {
    load_coords(cfname,unccfname);
    //    load_vels(vfname);
    istep = istep_restart + 1.0;
  }

  if( rgen_restart ) {
    generator.restart();
  }

  if( first_time ) {

    energy_eval();
    force_eval();

  }

  if( binsave ) {
    if( (first_time)&&(!rgen_restart) ) {
      record_traj(binfname,uncbinfname);
    }
    while( istep <= nstep ) {

      // compute pair separation list
      if ((inlup % nnlup) == 0) {
        if (neighborlist == 1) {
          update_neighbor_list();
        } else if (celllist == 1) {
          update_cell_list();
        }
	//	fprintf(stderr, "(%.0lf) neighbor list: (%d/%d)\n", istep, nnl_att, nnl_rep);
        inlup = 0;
      }
      inlup++;

      if (neighborlist == 1 || celllist == 1) {
        update_pair_list();
//	fprintf(stderr, "(%.0lf) pair list: (%d/%d)\n", istep, nil_att, nil_rep);
      }

      overdamped_iteration(incr);
      if( !(iup%nup) ) { // updates
	energy_eval();
	calculate_observables(incr);
        sprintf(oline,"%.0lf %f %f %f %f %f %f %f %d %f",
                istep,T,kinT,e_bnd,e_ang_ss,e_vdw_rr,rna_etot,
                Q,contct_nat,rgsq);
	out << oline << endl;
	iup = 0;
	record_traj(binfname,uncbinfname);
	save_coords(cfname,unccfname);
	save_vels(vfname);
	generator.save_state();
      }
      istep += 1.0;
      iup++;

    }
    out.close();
  }

  if( first_time ) first_time = 0;

  delete [] incr;

  return;

}

void update_pair_list() {

  using namespace std;

  // declare host variables
  double dx, dy, dz;
  double d2;
  unsigned int ibead, jbead, itype, jtype;
  double rcut, rcut2;

  nil_att = 0;
  nil_rep = 0;

  // declare device variables

  // should be native distance
  for (int i=0; i<nnl_att; i++) {

    ibead = ibead_neighbor_list_att[i];
    jbead = jbead_neighbor_list_att[i];
    itype = itype_neighbor_list_att[i];
    jtype = jtype_neighbor_list_att[i];

    dx = unc_pos[jbead-1].x - unc_pos[ibead-1].x;
    dy = unc_pos[jbead-1].y - unc_pos[ibead-1].y;
    dz = unc_pos[jbead-1].z - unc_pos[ibead-1].z;

    dx -= boxl*rnd(dx/boxl);
    dy -= boxl*rnd(dy/boxl);
    dz -= boxl*rnd(dz/boxl);

    d2 = dx*dx+dy*dy+dz*dz;

    rcut = 2.5*nl_lj_nat_pdb_dist[i];
    rcut2 = rcut*rcut;

    if (d2 < rcut2) {
      // add to interaction pair list
      ibead_pair_list_att[nil_att] = ibead;
      jbead_pair_list_att[nil_att] = jbead;
      itype_pair_list_att[nil_att] = itype;
      jtype_pair_list_att[nil_att] = jtype;
      pl_lj_nat_pdb_dist[nil_att] = nl_lj_nat_pdb_dist[i];
      pl_lj_nat_pdb_dist2[nil_att] = nl_lj_nat_pdb_dist2[i];
      pl_lj_nat_pdb_dist6[nil_att] = nl_lj_nat_pdb_dist6[i];
      pl_lj_nat_pdb_dist12[nil_att] = nl_lj_nat_pdb_dist12[i];
      nil_att++;
    }

  }

  for (int i=0; i<nnl_rep; i++) {

    ibead = ibead_neighbor_list_rep[i];
    jbead = jbead_neighbor_list_rep[i];
    itype = itype_neighbor_list_rep[i];
    jtype = jtype_neighbor_list_rep[i];

    dx = unc_pos[jbead-1].x - unc_pos[ibead-1].x;
    dy = unc_pos[jbead-1].y - unc_pos[ibead-1].y;
    dz = unc_pos[jbead-1].z - unc_pos[ibead-1].z;

    dx -= boxl*rnd(dx/boxl);
    dy -= boxl*rnd(dy/boxl);
    dz -= boxl*rnd(dz/boxl);

    d2 = dx*dx+dy*dy+dz*dz;

    rcut = 2.5*sigma_rep[itype][jtype];
    rcut2 = rcut*rcut;

    if (d2 < rcut2) {
      // add to interaction pair list
      ibead_pair_list_rep[nil_rep] = ibead;
      jbead_pair_list_rep[nil_rep] = jbead;
      itype_pair_list_rep[nil_rep] = itype;
      jtype_pair_list_rep[nil_rep] = jtype;
      nil_rep++;
    }
  }
}
