#include "hip/hip_runtime.h"
#include <math.h>
#include <cstdlib>
#include "neighbor_list.h"
#include "global.h"
#include <hip/hip_vector_types.h>
#include <stdio.h>
#include <stdlib.h>

#define SECTION_SIZE 1024

__device__ __constant__ double dev_sigma_rep[3][3] = {
	{0.0, 0.0, 0.0},
	{0.0, 3.8, 5.4},
	{0.0, 5.4, 7.0}
};

void update_neighbor_list() {

  double dx, dy, dz;
  double d2;
  int ibead, jbead, itype, jtype;
  double rcut, rcut2;

  nnl_att = 0;
  nnl_rep = 0;

  // calculations for native (attractiction) contacts
  for (int i=0; i<ncon_att; i++) {
    // record sigma for ibead and jbead
    ibead = ibead_lj_nat[i];
    jbead = jbead_lj_nat[i];

    // record type of bead for ibead and jbead
    itype = itype_lj_nat[i];
    jtype = jtype_lj_nat[i];
    
    // calculate distance in x, y, and z for ibead and jbead
    dx = unc_pos[jbead].x - unc_pos[ibead].x;
    dy = unc_pos[jbead].y - unc_pos[ibead].y;
    dz = unc_pos[jbead].z - unc_pos[ibead].z;

    // apply periodic boundary conditions to dx, dy, and dz
    dx -= boxl*rnd(dx/boxl);
    dy -= boxl*rnd(dy/boxl);
    dz -= boxl*rnd(dz/boxl);

    // compute square of distance between ibead and jbead
    d2 = dx*dx+dy*dy+dz*dz;

    /* 
    Compute the cutoff distance for the given bead
    This is based off of lj_nat_pdb_dist[i], which is the distance 
    from ibead to jbead in the resulting folded structure
    */
    rcut = 3.2*lj_nat_pdb_dist[i];

    // square cutoff distance, since sqrt(d2) is computationally expensive
    rcut2 = rcut*rcut;

    // checks if distance squared is less than the cutoff distance squared
    if (d2 < rcut2) {
      // add to neighbor list
      nnl_att++;
      // add pair to respective attraction neighbor lists
      ibead_neighbor_list_att[nnl_att] = ibead;
      jbead_neighbor_list_att[nnl_att] = jbead;
      
      // record type of each bead
      itype_neighbor_list_att[nnl_att] = itype;
      jtype_neighbor_list_att[nnl_att] = jtype;

      // record values, so that calculatons are not repeated (look-up table)
      nl_lj_nat_pdb_dist[nnl_att] = lj_nat_pdb_dist[i];
      nl_lj_nat_pdb_dist2[nnl_att] = lj_nat_pdb_dist2[i];
      nl_lj_nat_pdb_dist6[nnl_att] = lj_nat_pdb_dist6[i];
      nl_lj_nat_pdb_dist12[nnl_att] = lj_nat_pdb_dist12[i];
    }
  }
  printf("%d\n", nnl_att);
  fflush(stdout);


  // calculations for non-native (repulsive) contacts
  for (int i=0; i<ncon_rep; i++) {
    // record sigma for ibead and jbead
    ibead = ibead_lj_non_nat[i];
    jbead = jbead_lj_non_nat[i];

    // record type of bead for ibead and jbead
    itype = itype_lj_non_nat[i];
    jtype = jtype_lj_non_nat[i];

    // calculate distance in x, y, and z for ibead and jbead
    dx = unc_pos[jbead].x - unc_pos[ibead].x;
    dy = unc_pos[jbead].y - unc_pos[ibead].y;
    dz = unc_pos[jbead].z - unc_pos[ibead].z;

    // apply periodic boundary conditions to dx, dy, and dz
    dx -= boxl*rnd(dx/boxl);
    dy -= boxl*rnd(dy/boxl);
    dz -= boxl*rnd(dz/boxl);

    // compute square of distance between ibead and jbead
    d2 = dx*dx+dy*dy+dz*dz;

    /* 
    Compute the cutoff distance for the given bead
    This is based off of sigma_rep[itype][jtype],
    is based on the sigma for the types of ibead and jbead
    */
    rcut = 3.2*sigma_rep[itype][jtype];

    // square cutoff distance, since sqrt(d2) is computationally expensive
    rcut2 = rcut*rcut;

    // checks if distance squared is less than the cutoff distance squared
    if (d2 < rcut2) {
      // add to neighbor list
      nnl_rep++;

      // add pair to respective repulsive neighbor lists
      ibead_neighbor_list_rep[nnl_rep] = ibead;
      jbead_neighbor_list_rep[nnl_rep] = jbead;

      // record type of each bead
      itype_neighbor_list_rep[nnl_rep] = itype;
      jtype_neighbor_list_rep[nnl_rep] = jtype;
    }
  }
  printf("%d\n", nnl_rep);
  fflush(stdout);
}

void update_neighbor_list_gpu(){
 	// Declare N
	int N;
	
	// Set N
	N = ncon_att+1;
	
	// Declare value array
	int *value;
	value = (int *)malloc(N*sizeof(int));
	
	// Calculate binary list for att
	calculate_array_native(ibead_lj_nat, jbead_lj_nat, itype_lj_nat, jtype_lj_nat, unc_pos, lj_nat_pdb_dist, value, boxl, N);

    N--;

	// Compact ibead_neighbor_list_att
	nnl_att = compact(ibead_lj_nat+1, value+1, N, ibead_neighbor_list_att);
    printf("%d\n", nnl_att);
    fflush(stdout);

    /*
    for(int i = 0; i < N; i++){
        printf("%d\n", ibead_neighbor_list_att[i]);
    }
    fflush(stdout);
    exit(0);*/
	
	// Compact jbead_neighbor_list_att
	compact(jbead_lj_nat+1, value+1, N, jbead_neighbor_list_att);
	
	// Compact itype_neighbor_list_att
	compact(itype_lj_nat+1, value+1, N, itype_neighbor_list_att);
	
	// Compact jtype_neighbor_list_att
	compact(jtype_lj_nat+1, value+1, N, jtype_neighbor_list_att);
	
	// Compact nl_lj_nat_pdb_dist
	compact(lj_nat_pdb_dist+1, value+1, N, nl_lj_nat_pdb_dist);
	
	// Compact nl_lj_nat_pdb_dist2
	compact(lj_nat_pdb_dist2+1, value+1, N, nl_lj_nat_pdb_dist2);
	
	// Compact nl_lj_nat_pdb_dist6
	compact(lj_nat_pdb_dist6+1, value+1, N, nl_lj_nat_pdb_dist6);
	
	// Compact nl_lj_nat_pdb_dist12
	compact(lj_nat_pdb_dist12+1, value+1, N, nl_lj_nat_pdb_dist12);
	
	// Free value memory
	free(value);
	
	
	/**********************************
	 *								  *
	 * End of Attractive Calculations *
	 *								  *
	 **********************************/
	
	
	// Set N
	N = ncon_rep+1;
	
	// Declare value array
	value = (int *)malloc(N*sizeof(int));
	
	// Calculate binary list for rep
	calculate_array_non_native(ibead_lj_non_nat, jbead_lj_non_nat, itype_lj_non_nat, jtype_lj_non_nat, unc_pos, value, boxl, N);
	
    N--;

	// Compact ibead_neighbor_list_rep
	nnl_rep = compact(ibead_lj_non_nat+1, value+1, N, ibead_neighbor_list_rep);
    printf("%d\n", nnl_rep);
    fflush(stdout);
	
	// Compact jbead_neighbor_list_rep
	compact(jbead_lj_non_nat+1, value+1, N, jbead_neighbor_list_rep);
	
	// Compact itype_neighbor_list_rep
	compact(itype_lj_non_nat+1, value+1, N, itype_neighbor_list_rep);
	
	// Compact jtype_neighbor_list_rep
	compact(itype_lj_non_nat+1, value+1, N, itype_neighbor_list_rep);

    free(value);
}

void calculate_array_native(int *ibead_lj_nat, int *jbead_lj_nat, int *itype_lj_nat, int *jtype_lj_nat, float3 *unc_pos, double *lj_nat_pdb_dist, 
                            int *value, int boxl, int N){
							
	// Calculate array sizes
	int size_int = N*sizeof(int);
	int size_double = N*sizeof(double);
	int size_float3 = (nbead+1)*sizeof(float3);
	
	// Declare device pointers
	int *dev_ibead_lj_nat;
	int *dev_jbead_lj_nat;
	int *dev_itype_lj_nat;
	int *dev_jtype_lj_nat;
	float3 *dev_unc_pos;
	double *dev_lj_nat_pdb_dist; 
	int *dev_value;
	
	// Allocate device arrays
	hipMalloc((void **)&dev_ibead_lj_nat, size_int);	
	hipMalloc((void **)&dev_jbead_lj_nat, size_int);
	hipMalloc((void **)&dev_itype_lj_nat, size_int);
	hipMalloc((void **)&dev_jtype_lj_nat, size_int);
	hipMalloc((void **)&dev_unc_pos, size_float3);
	hipMalloc((void **)&dev_lj_nat_pdb_dist, size_double);
	hipMalloc((void **)&dev_value, size_int);
	
	// Copy host arrays to device arrays
	hipMemcpy(dev_ibead_lj_nat, ibead_lj_nat, size_int, hipMemcpyHostToDevice);
	hipMemcpy(dev_jbead_lj_nat, jbead_lj_nat, size_int, hipMemcpyHostToDevice);
	hipMemcpy(dev_itype_lj_nat, itype_lj_nat, size_int, hipMemcpyHostToDevice);
	hipMemcpy(dev_jtype_lj_nat, jtype_lj_nat, size_int, hipMemcpyHostToDevice);
	hipMemcpy(dev_unc_pos, unc_pos, size_float3, hipMemcpyHostToDevice);
	hipMemcpy(dev_lj_nat_pdb_dist, lj_nat_pdb_dist, size_double, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
	
	// Calculate block/thread count
	int threads = (int)min(N, SECTION_SIZE);
    int blocks = (int)ceil(1.0*N/SECTION_SIZE);

    //dummy<<<blocks, threads>>>(dev_ibead_lj_nat, dev_jbead_lj_nat, dev_itype_lj_nat, dev_jtype_lj_nat, dev_unc_pos, dev_lj_nat_pdb_dist, dev_value, boxl, N, nbead);
    
    /*
    for(int i = 0; i < N; i++){
        printf("i: %d  j: %d\n",ibead_lj_nat[i],jbead_lj_nat[i]);
        float3 a = unc_pos[ibead_lj_nat[i]];
        printf("%d: x=%f, y=%f, z=%f\n", ibead_lj_nat[i], a.x, a.y, a.z);
        
        a = unc_pos[jbead_lj_nat[i]];
        printf("%d: x=%f, y=%f, z=%f\n", jbead_lj_nat[i], a.x, a.y, a.z);
    }
    
    int *test_ibead_lj_nat = (int *)malloc(size_int);
    int *test_jbead_lj_nat = (int *)malloc(size_int);
    float3 *test_unc_pos = (float3 *)malloc(size_float3);
    
    hipMemcpy(test_ibead_lj_nat, dev_ibead_lj_nat, size_int, hipMemcpyDeviceToHost);
	hipMemcpy(test_jbead_lj_nat, dev_jbead_lj_nat, size_int, hipMemcpyDeviceToHost);
    hipMemcpy(test_unc_pos, dev_unc_pos, size_float3, hipMemcpyDeviceToHost);
    
    for(int i = 0; i < N; i++){
        if(i > 0 && i < N){
            if(ibead_lj_nat[i] != test_ibead_lj_nat[i] || jbead_lj_nat[i] != test_jbead_lj_nat[i] || unc_pos[ibead_lj_nat[i]].x != test_unc_pos[test_ibead_lj_nat[i]].x ||
            unc_pos[ibead_lj_nat[i]].y != test_unc_pos[test_ibead_lj_nat[i]].y || unc_pos[ibead_lj_nat[i]].z != test_unc_pos[test_ibead_lj_nat[i]].z || 
            unc_pos[jbead_lj_nat[i]].x != test_unc_pos[test_jbead_lj_nat[i]].x || unc_pos[jbead_lj_nat[i]].y != test_unc_pos[test_jbead_lj_nat[i]].y ||
            unc_pos[jbead_lj_nat[i]].z != test_unc_pos[test_jbead_lj_nat[i]].z){
                printf("i: %d  j: %d\n",ibead_lj_nat[i],jbead_lj_nat[i]);
                float3 a = unc_pos[ibead_lj_nat[i]];
                printf("%d: x=%f, y=%f, z=%f\n", ibead_lj_nat[i], a.x, a.y, a.z);
                
                a = unc_pos[jbead_lj_nat[i]];
                printf("%d: x=%f, y=%f, z=%f\n", jbead_lj_nat[i], a.x, a.y, a.z);
                
                printf("test_i: %d  test_j: %d\n",test_ibead_lj_nat[i],test_jbead_lj_nat[i]);
                a = test_unc_pos[test_ibead_lj_nat[i]];
                printf("test_%d: x=%f, y=%f, z=%f\n", test_ibead_lj_nat[i], a.x, a.y, a.z);
                
                a = test_unc_pos[test_jbead_lj_nat[i]];
                printf("test_%d: x=%f, y=%f, z=%f\n", test_jbead_lj_nat[i], a.x, a.y, a.z);
            }
        }
    }*/
	
	// Compute binary array
	array_native_kernel<<<blocks, threads>>>(dev_ibead_lj_nat, dev_jbead_lj_nat, dev_itype_lj_nat, dev_jtype_lj_nat, dev_unc_pos, dev_lj_nat_pdb_dist, dev_value, boxl, N);

    // Sync device
    hipDeviceSynchronize();

	// Copy device array to host array
	hipMemcpy(value, dev_value, size_int, hipMemcpyDeviceToHost);
	
    hipDeviceSynchronize();

	// Free GPU memory
	hipFree(dev_ibead_lj_nat);
	hipFree(dev_jbead_lj_nat);
	hipFree(dev_itype_lj_nat);
	hipFree(dev_jtype_lj_nat);
	hipFree(dev_unc_pos);
	hipFree(dev_lj_nat_pdb_dist);
	hipFree(dev_value);
}

__global__ void array_native_kernel(int *dev_ibead_lj_nat, int *dev_jbead_lj_nat, int *dev_itype_lj_nat, int *dev_jtype_lj_nat, float3 *dev_unc_pos, double *dev_lj_nat_pdb_dist, 
                            int *dev_value, int boxl, int N){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i > 0 && i < N){
    double dx, dy, dz;
    double d2;
    int ibead, jbead, itype, jtype;
    double rcut, rcut2;

    // record sigma for ibead and jbead
    ibead = dev_ibead_lj_nat[i];

    jbead = dev_jbead_lj_nat[i];

    // record type of bead for ibead and jbead
    itype = dev_itype_lj_nat[i];

    jtype = dev_jtype_lj_nat[i];
    
    // calculate distance in x, y, and z for ibead and jbead
    dx = dev_unc_pos[jbead].x - dev_unc_pos[ibead].x;

    dy = dev_unc_pos[jbead].y - dev_unc_pos[ibead].y;

    dz = dev_unc_pos[jbead].z - dev_unc_pos[ibead].z;

    // apply periodic boundary conditions to dx, dy, and dz
    //dx -= boxl*rnd(dx/boxl);
    double rnd_value;

    rnd_value = ( ((dx/boxl)>0) ? std::floor((dx/boxl)+0.5) : std::ceil((dx/boxl)-0.5) );
    dx -= boxl*rnd_value;

    //dy -= boxl*rnd(dy/boxl);
    rnd_value = ( ((dy/boxl)>0) ? std::floor((dy/boxl)+0.5) : std::ceil((dy/boxl)-0.5) );
    dy -= boxl*rnd_value;

    //dz -= boxl*rnd(dz/boxl);
    rnd_value = ( ((dz/boxl)>0) ? std::floor((dz/boxl)+0.5) : std::ceil((dz/boxl)-0.5) );
    dz -= boxl*rnd_value;

    // compute square of distance between ibead and jbead
    d2 = dx*dx+dy*dy+dz*dz;

    /* 
    Compute the cutoff distance for the given bead
    This is based off of lj_nat_pdb_dist[i], which is the distance 
    from ibead to jbead in the resulting folded structure
    */
    rcut = 3.2*dev_lj_nat_pdb_dist[i];

    // square cutoff distance, since sqrt(d2) is computationally expensive
    rcut2 = rcut*rcut;

    if(d2 < rcut2){
      dev_value[i] = 1;
    }else{
      dev_value[i] = 0;
    }
  }
}

void calculate_array_non_native(int *ibead_lj_non_nat, int *jbead_lj_non_nat, int *itype_lj_non_nat, int *jtype_lj_non_nat, float3 *unc_pos,
                            int *value, int boxl, int N){
							
	// Calculate array sizes
	int size_int = N*sizeof(int);
	int size_double = N*sizeof(double);
	int size_float3 = (nbead+1)*sizeof(float3);
	
	// Declare device pointers
	int *dev_ibead_lj_non_nat;
	int *dev_jbead_lj_non_nat;
	int *dev_itype_lj_non_nat;
	int *dev_jtype_lj_non_nat;
	float3 *dev_unc_pos; 
	int *dev_value;
	
	// Allocate device arrays
	hipMalloc((void **)&dev_ibead_lj_non_nat, size_int);	
	hipMalloc((void **)&dev_jbead_lj_non_nat, size_int);
	hipMalloc((void **)&dev_itype_lj_non_nat, size_int);
	hipMalloc((void **)&dev_jtype_lj_non_nat, size_int);
	hipMalloc((void **)&dev_unc_pos, size_float3);
	hipMalloc((void **)&dev_value, size_int);
	
	// Copy host arrays to device arrays
	hipMemcpy(dev_ibead_lj_non_nat, ibead_lj_non_nat, size_int, hipMemcpyHostToDevice);
	hipMemcpy(dev_jbead_lj_non_nat, jbead_lj_non_nat, size_int, hipMemcpyHostToDevice);
	hipMemcpy(dev_itype_lj_non_nat, itype_lj_non_nat, size_int, hipMemcpyHostToDevice);
	hipMemcpy(dev_jtype_lj_non_nat, jtype_lj_non_nat, size_int, hipMemcpyHostToDevice);
	hipMemcpy(dev_unc_pos, unc_pos, size_float3, hipMemcpyHostToDevice);
	hipMemcpy(dev_value, value, size_int, hipMemcpyHostToDevice);
	
	// Calculate block/thread count
	int threads = (int)min(N, SECTION_SIZE);
    int blocks = (int)ceil(1.0*N/SECTION_SIZE);
	
	// Compute binary array
	array_non_native_kernel<<<blocks, threads>>>(dev_ibead_lj_non_nat, dev_jbead_lj_non_nat, dev_itype_lj_non_nat, dev_jtype_lj_non_nat, 
                                                dev_unc_pos, dev_value, boxl, N);
    /*
    hipDeviceSynchronize();

    // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }else{
        printf("Success\n");
        exit(0);
    }*/
	
    // Sync device
    hipDeviceSynchronize();

	// Copy device array to host array
	hipMemcpy(value, dev_value, size_int, hipMemcpyDeviceToHost);

	// Free GPU memory
	hipFree(dev_ibead_lj_non_nat);
	hipFree(dev_jbead_lj_non_nat);
	hipFree(dev_itype_lj_non_nat);
	hipFree(dev_jtype_lj_non_nat);
	hipFree(dev_unc_pos);
	hipFree(dev_value);
}

__global__ void array_non_native_kernel(int *dev_ibead_lj_non_nat, int *dev_jbead_lj_non_nat, int *dev_itype_lj_non_nat, int *dev_jtype_lj_non_nat, 
                                        float3 *dev_unc_pos, int *dev_value, int boxl, int N){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i > 0 && i < N){
    double dx, dy, dz;
    double d2;
    int ibead, jbead, itype, jtype;
    double rcut, rcut2;

    // record sigma for ibead and jbead
    ibead = dev_ibead_lj_non_nat[i];
    jbead = dev_jbead_lj_non_nat[i];

    // record type of bead for ibead and jbead
    itype = dev_itype_lj_non_nat[i];
    jtype = dev_jtype_lj_non_nat[i];
    
    // calculate distance in x, y, and z for ibead and jbead
    dx = dev_unc_pos[jbead].x - dev_unc_pos[ibead].x;
    dy = dev_unc_pos[jbead].y - dev_unc_pos[ibead].y;
    dz = dev_unc_pos[jbead].z - dev_unc_pos[ibead].z;

    // apply periodic boundary conditions to dx, dy, and dz
    //dx -= boxl*rnd(dx/boxl);
    double rnd_value;

    rnd_value = ( ((dx/boxl)>0) ? std::floor((dx/boxl)+0.5) : std::ceil((dx/boxl)-0.5) );
    dx -= boxl*rnd_value;

    //dy -= boxl*rnd(dy/boxl);
    rnd_value = ( ((dy/boxl)>0) ? std::floor((dy/boxl)+0.5) : std::ceil((dy/boxl)-0.5) );
    dy -= boxl*rnd_value;

    //dz -= boxl*rnd(dz/boxl);
    rnd_value = ( ((dz/boxl)>0) ? std::floor((dz/boxl)+0.5) : std::ceil((dz/boxl)-0.5) );
    dz -= boxl*rnd_value;

    // compute square of distance between ibead and jbead
    d2 = dx*dx+dy*dy+dz*dz;

    /* 
    Compute the cutoff distance for the given bead
    This is based off of lj_nat_pdb_dist[i], which is the distance 
    from ibead to jbead in the resulting folded structure
    */
	// May need to change to dev_sigma_rep[N*itype + jtype]
    rcut = 3.2*dev_sigma_rep[itype][jtype];

    // square cutoff distance, since sqrt(d2) is computationally expensive
    rcut2 = rcut*rcut;

    if(d2 < rcut2){
      dev_value[i] = 1;
    }else{
      dev_value[i] = 0;
    }
  }
}

/*
 * Function: compact
 * -----------------
 *  Finds points in index with a 1 in value and stores them
 *
 *  index: array of indices to check
 *  value: binary value indicating if the corresponding index value is true (1) or false (0)
 *  N: number of elements in index and value
 *  result: pointer where compacted array is stored
 *
 *  Returns: arrSize, the size of the compacted array
 *           Note: result is modified in-place
 */

int compact(int *index, int *value, int N, int *&result){
    // Declare pointers for dev_output and dev_value arrays
    int *dev_output;
    int *dev_value;

    // Calculate array size
    int size = N * sizeof(int);

    // Allocate dev_value and dev_output arrays
    hipMalloc((void**)&dev_value, size);
    hipMalloc((void**)&dev_output, size);
 
    // Copy data from value array to device (dev_value)
    hipMemcpy(dev_value, value, size, hipMemcpyHostToDevice);

    // Perform hierarchical Kogge-Stone scan on dev_value array and store result in dev_output
    hier_ks_scan(dev_value, dev_output, N, 0);

    // Copy size of compacted array from device to host and store in arrSize
    /* 
     * TODO: If the entire array has 1 as the value, an exclusive scan will have N-1 as the last value in the array.
     * However, allocating an array with N-1 entries will not store all N values from the index array.
     * Change code to determine when we need to increment arrSize and when we don't.
     * Options include:
     *  1) Changing the hierarchical scan kernel to determine if the final value in the value array is 1
     *  2) Checking to see if the final value is 1 in the value array
     * Option 2 was selected, but please double-check this approach
     */ 
    int arrSize;
    hipMemcpy(&arrSize, &dev_output[N-1], sizeof(int), hipMemcpyDeviceToHost); 

    // Increment arrSize by 1 if needed
    if(value[N-1]){
        arrSize++;
    }

    // Declare and allocate dev_result array to store compacted indices on device (on GPU)
    int *dev_result;
    hipMalloc((void**)&dev_result, arrSize*sizeof(int));

    // Declare and allocate dev_index to store indecies (on GPU)
    int *dev_index;
    hipMalloc((void**)&dev_index, size);

    // Copy indices from host to device
    hipMemcpy(dev_index, index, size, hipMemcpyHostToDevice);

    /* Calculate number of threads and blocks to use for copying
     * If N < SECTION_SIZE (max # of threads per block), use N threads per block. Else, use SECTION_SIZE threads per block
     * Divides number of elements in array by SECTION_SIZE and rounds up, ensuring it uses the minimum number of blocks required
     */
    int threads = (int)min(N, SECTION_SIZE);
    int blocks = (int)ceil(1.0*N/SECTION_SIZE);

    // Kernel to copy elements from dev_index to dev_output if their corresponding dev_value is 1
    copyElements<<<blocks, threads>>>(dev_index, dev_value, dev_output, dev_result, N);
    
    // Sync device to ensure GPU computation is finished before proceeding
    hipDeviceSynchronize();

    // Allocate result array on host
    free(result);
    result = (int *)malloc(arrSize*sizeof(int));

    // Copy dev_result (compacted array of indices in GPU) to result array on host
    hipMemcpy(result, dev_result, arrSize*sizeof(int), hipMemcpyDeviceToHost); 
    
    // Free device memory
    hipFree(dev_result); 
    hipFree(dev_index);
    hipFree(dev_value);
    hipFree(dev_output);

    return arrSize;
}

/*
 * Function: copyElements
 * -----------------
 *  Copys values marked true (1) from index array to result array
 *
 *  dev_index: array of indices to check (on GPU)
 *  dev_value: binary value indicating if the corresponding dev_index value is true (1) or false (0) (on GPU)
 *  N: number of elements in dev_index and dev_value
 *  dev_result: pointer where compacted array is stored (on GPU)
 */

__global__ void copyElements(int *dev_index, int *dev_value, int *dev_output, int *dev_result, int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x+1;
    if(dev_value[i] && i < N){
        dev_result[dev_output[i]-1] = dev_index[i];
    }
    return;
}

int compact(double *index, int *value, int N, double *&result){
    // Declare pointers for dev_output and dev_value arrays
    int *dev_output;
    int *dev_value;

    // Calculate array size
    int size = N * sizeof(int);

    // Allocate dev_value and dev_output arrays
    hipMalloc((void**)&dev_value, size);
    hipMalloc((void**)&dev_output, size);
 
    // Copy data from value array to device (dev_value)
    hipMemcpy(dev_value, value, size, hipMemcpyHostToDevice);

    // Perform hierarchical Kogge-Stone scan on dev_value array and store result in dev_output
    hier_ks_scan(dev_value, dev_output, N, 0);

    // Copy size of compacted array from device to host and store in arrSize
    /* 
     * TODO: If the entire array has 1 as the value, an exclusive scan will have N-1 as the last value in the array.
     * However, allocating an array with N-1 entries will not store all N values from the index array.
     * Change code to determine when we need to increment arrSize and when we don't.
     * Options include:
     *  1) Changing the hierarchical scan kernel to determine if the final value in the value array is 1
     *  2) Checking to see if the final value is 1 in the value array
     * Option 2 was selected, but please double-check this approach
     */ 
    int arrSize;
    hipMemcpy(&arrSize, &dev_output[N-1], sizeof(int), hipMemcpyDeviceToHost); 

    // Increment arrSize by 1 if needed
    if(value[N-1]){
        arrSize++;
    }

    // Declare and allocate dev_result array to store compacted indices on device (on GPU)
    double *dev_result;
    hipMalloc((void**)&dev_result, arrSize*sizeof(double));

    // Declare and allocate dev_index to store indecies (on GPU)
    double *dev_index;
    hipMalloc((void**)&dev_index, N*sizeof(double));

    // Copy indices from host to device
    hipMemcpy(dev_index, index, N*sizeof(double), hipMemcpyHostToDevice);

    /* Calculate number of threads and blocks to use for copying
     * If N < SECTION_SIZE (max # of threads per block), use N threads per block. Else, use SECTION_SIZE threads per block
     * Divides number of elements in array by SECTION_SIZE and rounds up, ensuring it uses the minimum number of blocks required
     */
    int threads = (int)min(N, SECTION_SIZE);
    int blocks = (int)ceil(1.0*N/SECTION_SIZE);

    // Kernel to copy elements from dev_index to dev_output if their corresponding dev_value is 1
    copyElements<<<blocks, threads>>>(dev_index, dev_value, dev_output, dev_result, N);
    
    // Sync device to ensure GPU computation is finished before proceeding
    hipDeviceSynchronize();

    // Allocate result array on host
    free(result);
    result = (double *)malloc(arrSize*sizeof(double));

    // Copy dev_result (compacted array of indices in GPU) to result array on host
    hipMemcpy(result, dev_result, arrSize*sizeof(double), hipMemcpyDeviceToHost); 
    
    // Free device memory
    hipFree(dev_result); 
    hipFree(dev_index);
    hipFree(dev_value);
    hipFree(dev_output);

    return arrSize;
}

/*
 * Function: copyElements
 * -----------------
 *  Copys values marked true (1) from index array to result array
 *
 *  dev_index: array of indices to check (on GPU)
 *  dev_value: binary value indicating if the corresponding dev_index value is true (1) or false (0) (on GPU)
 *  N: number of elements in dev_index and dev_value
 *  dev_result: pointer where compacted array is stored (on GPU)
 */

__global__ void copyElements(double *dev_index, int *dev_value, int *dev_output, double *dev_result, int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x+1;
    if(dev_value[i] && i < N){
        dev_result[dev_output[i]-1] = dev_index[i];
    }
    return;
}

/*
 * Function: hier_ks_scan
 * -----------------
 *  
 *
 *  dev_index: array of indices to check (on GPU)
 *  dev_value: binary value indicating if the corresponding dev_index value is true (1) or false (0) (on GPU)
 *  N: number of elements in dev_index and dev_value
 *  dev_result: pointer where compacted array is stored (on GPU)
 */

void hier_ks_scan(int *dev_X, int *dev_Y, int N, int re){
    if(N <= SECTION_SIZE){
        ksScanInc<<<1, N>>>(dev_X, dev_Y, N);

        hipDeviceSynchronize();

        return;
    }else{
        int threads = (int)min(N, SECTION_SIZE);
        int blocks = (int)ceil(1.0*N/SECTION_SIZE);

        int *dev_S;
        hipMalloc((void**)&dev_S, (int)ceil(1.0*N/SECTION_SIZE) * sizeof(int));
        
        ksScanAuxInc<<<blocks, threads>>>(dev_X, dev_Y, N, dev_S);
        hipDeviceSynchronize();

        hier_ks_scan(dev_S, dev_S, (int)ceil(1.0*N/SECTION_SIZE), 1);
        hipDeviceSynchronize();
        
        sumIt<<<blocks, threads>>>(dev_Y, dev_S, N);
        hipDeviceSynchronize();

        hipFree(dev_S);

        return;
    }
}

__global__ void ksScanAuxExc (int *X, int *Y, int InputSize, int *S) {
    int val;
    
    __shared__ int XY[SECTION_SIZE];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < InputSize && threadIdx.x != 0){
        XY[threadIdx.x] = X[i-1];
    }else{
        XY[threadIdx.x] = 0;
    }

    for(unsigned int stride = 1; stride < blockDim.x; stride *=2){
        __syncthreads();
        if(threadIdx.x >= stride){
            val = XY[threadIdx.x - stride];
        }
        __syncthreads();
        if(threadIdx.x >= stride){
            XY[threadIdx.x] += val;
        }
    }

    __syncthreads();
    if(i < InputSize){
        Y[i] = XY[threadIdx.x];
    }
    
    __syncthreads();
    if(threadIdx.x == 0){
        S[blockIdx.x] = XY[SECTION_SIZE-1];
    }
}

__global__ void ksScanAuxInc (int *X, int *Y, int InputSize, int *S) {
    int val;
    
    __shared__ int XY[SECTION_SIZE];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < InputSize){
        XY[threadIdx.x] = X[i];
    }

    for(unsigned int stride = 1; stride < blockDim.x; stride *=2){
        __syncthreads();
        if(threadIdx.x >= stride){
            val = XY[threadIdx.x - stride];
        }
        __syncthreads();
        if(threadIdx.x >= stride){
            XY[threadIdx.x] += val;
        }
    }

    __syncthreads();
    if(i < InputSize){
        Y[i] = XY[threadIdx.x];
    }
    
    __syncthreads();
    if(threadIdx.x == 0){
        S[blockIdx.x] = XY[SECTION_SIZE-1];
    }
}

__global__ void ksScanExc (int *X, int *Y, int InputSize) {
    int val;
    
    __shared__ int XY[SECTION_SIZE];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(i < InputSize && threadIdx.x != 0){
        XY[threadIdx.x] = X[i-1];
    }else{
        XY[threadIdx.x] = 0;
    }

    for(unsigned int stride = 1; stride < blockDim.x; stride *=2){
        __syncthreads();
        if(threadIdx.x >= stride){
            val = XY[threadIdx.x - stride];
        }
        __syncthreads();
        if(threadIdx.x >= stride){
            XY[threadIdx.x] += val;
        }
    }

    __syncthreads();
    if(i < InputSize){
        Y[i] = XY[threadIdx.x];
    }
}

__global__ void ksScanInc (int *X, int *Y, int InputSize) {
    int val;
    
    __shared__ int XY[SECTION_SIZE];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(i < InputSize){
        XY[threadIdx.x] = X[i];
    }

    for(unsigned int stride = 1; stride < blockDim.x; stride *=2){
        __syncthreads();
        if(threadIdx.x >= stride){
            val = XY[threadIdx.x - stride];
        }
        __syncthreads();
        if(threadIdx.x >= stride){
            XY[threadIdx.x] += val;
        }
    }

    __syncthreads();
    if(i < InputSize){
        Y[i] = XY[threadIdx.x];
    }
}

__global__ void sumIt (int *Y, int *S, int InputSize) {
    if(blockIdx.x > 0){
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if(i < InputSize){
            Y[i] += S[blockIdx.x-1];
        }
    }
}

/*
__global__ void dummy(int *dev_ibead_lj_nat, int *dev_jbead_lj_nat, int *dev_itype_lj_nat, int *dev_jtype_lj_nat, float3 *dev_unc_pos, double *dev_lj_nat_pdb_dist, 
                            int *&dev_value, int boxl, int N, int nbead){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i > 0 && i < N){
    double dx, dy, dz;
    double d2;
    int ibead, jbead, itype, jtype;
    double rcut, rcut2;

    if(i >= N){
        printf("%d\n", i);
    }

    // record sigma for ibead and jbead
    ibead = dev_ibead_lj_nat[i];
    //printf("dev_ibead_lj_nat[%d]", i);
    
    jbead = dev_jbead_lj_nat[i];
    //printf("dev_jbead_lj_nat[%d]", i);
    
    // record type of bead for ibead and jbead
    itype = dev_itype_lj_nat[i];
    //printf("dev_itype_lj_nat[%d]", i);
    
    jtype = dev_jtype_lj_nat[i];
    //printf("dev_jtype_lj_nat[%d]", i);
    
    
    if(ibead > nbead+1){
        printf("ibead: %d\n", i);
    }else if(jbead > nbead+1){
        printf("jbead: %d\n", i);
    }
    
    
    // calculate distance in x, y, and z for ibead and jbead
    dx = dev_unc_pos[jbead].x - dev_unc_pos[ibead].x;
    printf("dev_unc_pos[%d].x - dev_unc_pos[%d].x", jbead, ibead);
    
    dy = dev_unc_pos[jbead].y - dev_unc_pos[ibead].y;
    printf("dev_unc_pos[%d].y - dev_unc_pos[%d].y", jbead, ibead);

    dz = dev_unc_pos[jbead].z - dev_unc_pos[ibead].z;
    printf("dev_unc_pos[%d].z - dev_unc_pos[%d].z", jbead, ibead);

    // apply periodic boundary conditions to dx, dy, and dz
    //dx -= boxl*rnd(dx/boxl);
    double rnd_value;

    rnd_value = ( ((dx/boxl)>0) ? std::floor((dx/boxl)+0.5) : std::ceil((dx/boxl)-0.5) );
    dx -= boxl*rnd_value;

    //dy -= boxl*rnd(dy/boxl);
    rnd_value = ( ((dy/boxl)>0) ? std::floor((dy/boxl)+0.5) : std::ceil((dy/boxl)-0.5) );
    dy -= boxl*rnd_value;

    //dz -= boxl*rnd(dz/boxl);
    rnd_value = ( ((dz/boxl)>0) ? std::floor((dz/boxl)+0.5) : std::ceil((dz/boxl)-0.5) );
    dz -= boxl*rnd_value;

    // compute square of distance between ibead and jbead
    d2 = dx*dx+dy*dy+dz*dz;

    rcut = 3.2*dev_lj_nat_pdb_dist[i];
    printf("dev_lj_nat_pdb_dist[%d]", i);

    // square cutoff distance, since sqrt(d2) is computationally expensive
    rcut2 = rcut*rcut;

    if(d2 < rcut2){
      dev_value[i] = 1;
    }else{
      dev_value[i] = 0;
    }
    
  }
}
*/